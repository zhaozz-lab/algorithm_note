
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <vector>
#include <iostream>
#include <algorithm>
#include<chrono>
#include <thread>
using namespace std;

__global__ void myfirstkernel(void) {
	printf("Hello CUDA \n");

}


__global__ void gpuAdd(int d_a, int d_b, int* d_c) {
	*d_c = d_a + d_b;
}


bool candy(std::vector<int>& ratings) {
	int size = ratings.size();
	if (size < 2)
	{
		return size;
	}
	std::vector<int> num(size, 1);
	for (int i = 1; i < size; i++) {
		if (ratings[i] > ratings[i - 1])
			num[i] = num[i-1]+1;
	}

	for (int i = size - 1; i > 0; --i) {
		if (ratings[i] < ratings[i - 1])
			num[i - 1] = std::max(num[i - 1], num[i] + 1);
	}
    
	for (auto i : num) {
		std::cout << i << std::endl;
	}
	return true;
}


bool testcandey() {

	std::vector<int> rating;
	rating.reserve(5);
	rating.push_back(1);
	rating.push_back(0);
	rating.push_back(2);
	rating.push_back(3);
	rating.push_back(4);
	candy(rating);
	return true;
}


vector<int> twosum(vector<int>& nums, int target) {
	int l = 0, r = nums.size()-1, sum;
	while (l<r)
	{
		sum = nums[l] + nums[r];
		if (sum == target)
			break;
		if (sum<target)
		{
			l++;
		}
		else
		{
			r--;
		}
	}
	return vector<int>{l, r};
}


bool testtwosum() {
	vector<int> numbers({ 2,7,11,15 });
	int target = 9;
	vector<int> results = twosum(numbers, target);
	for (auto result : results)
		cout << "the two sum is " << result << endl;

	return true;
}


int eraseOverlapIntervals(vector<vector<int>>& intervals) {
	if (intervals.empty()) {
		return 1;
	}
	int n = intervals.size();
	sort(intervals.begin(), intervals.end(), [](vector<int> a, vector<int> b) {return a[1] < b[1]; });
	int total = 0, prev = intervals[0][1];
	for (int i = 1; i < intervals.size(); ++i)
	{
		if (intervals[i][0] < prev) {
			++total;
		}
		else
		{
			prev = intervals[i][1];
		}
	}
	return total;
}


bool testeraseOverlapIntervals(){
	std::vector<vector<int>> intervals;
	intervals.push_back({1,3});
	intervals.push_back({2,3});
	intervals.push_back({2,4});
	intervals.push_back({4,6});
	cout << "the total is " << eraseOverlapIntervals(intervals) << endl;
	return true;
}


vector<int> merge(vector<int>& nums1,int m, vector<int>& nums2,int n) {
	int pos = m + n-1;
	vector<int> results;
	results.reserve(pos);
	m = m - 1;
	n = n - 1;
	while (m>=0 && n>=0)
	{
		if (nums1[m] > nums2[n])
		{
			results.push_back(nums1[m]);
			m = m - 1;
		}
		else
		{
			results.push_back(nums2[n]);
			n = n - 1;
		}

	}
	while (m>=0)
	{
		results.push_back(nums1[m]);
		m = m - 1;
	}

	while (n >= 0)
	{
		results.push_back(nums2[n]);
		n = n - 1;
	}
	return results;
}

void testmerge() {
	vector<int> nums1({ 1,2,3 });
	vector<int> nums2({ 2,5,6 });
	vector<int> results = merge(nums1,nums1.size(),nums2,nums2.size());
	for (auto result : results)
		cout << "the merge is " << result << endl;
	
}


struct Node
{
	int val;
	Node* next;
	Node(int x) :val(x), next(nullptr) {
	}
};


Node* detectCircle(Node* head) {
	Node* fast = head, * slow = head;
	do {
		if (!fast || !fast->next) return nullptr;
		fast = fast->next->next;
		slow = slow->next;
	} while (fast != slow);

	fast = head;
	while (fast!=slow)
	{
		fast = fast->next;
		slow = slow->next;
	}
	return fast;
}


void testdetectCircle() {

	Node* head = new Node(22);
	Node* p=head;
	for (int i = 0; i < 5; i++) {
		Node* temp = new Node(i);
		p->next = temp;
		p = temp;
	}
	p->next = head->next->next;
	p = head;
	
	Node* pjoin = detectCircle(head);
	cout << pjoin->val << endl;
	//while (p)
	//{
	//	cout << p->val << endl;
	//	p = p->next;
	//}
	

}


string minWindow(string S, string T) {

	vector<int> chars(128, 0);
	vector<bool> flag(128, false);
	//先统计T中的字符情况
	for (int i = 0; i < T.size(); i++) {
		flag[T[i]] = true;
		++chars[T[i]];
	}

	//移动滑动窗口，不断更改统计数据
	int cnt = 0, l = 0, min_l = 0, min_size = S.size() + 1;
	for (int r = 0; r < S.size(); ++r) {
		if (flag[S[r]]) {
			if (--chars[S[r]] >= 0) {
				++cnt;
			}
		}
		
		//若目前滑动窗口已包含T中全部字符，
		//则尝试将l右移，在不影响结果的情况下获得最短字符串
		
		while (cnt == T.size())
		{
			if (r - l + 1 < min_size) {
				min_l = l;
				min_size = r - l + 1;
			}
			if (flag[S[l]] && ++chars[S[l]] > 0) {
				--cnt;
			}
			++l;
		}
		

	}
	return  min_size > S.size() ? "" : S.substr(min_l, min_size);
}


void testminwindow() {

	string s = "ADOBECODEBANC";
	string t = "ABC";
	string result = minWindow(s, t);
	cout << "the result is " << result << endl;
}


int mysqrt(int a) {
	if (a == 0)
		return a;
	int l = 0, r = a, mid,sqrt;
	while (l<r)
	{
		mid = l + 0.5 * (r - l);
		sqrt = a / mid;
		if (sqrt == mid)
			return mid;
		else if (sqrt > mid)
			l = mid;
		else
			r = mid;
	}
	return r;
}


void testsqrt()
{

	//int a = 100;
	int result = mysqrt(100);
	cout << "the result is " << result << endl;
}


//区间查找
int lower_bound(vector<int>& nums, int target) {
	int l = 0, r = nums.size(), mid;
	while (l<r)
	{
		mid = (l + r) / 2;
		if (nums[mid]>=target)
		{
			r = mid;
		}
		else
		{
			l = mid+1;
		}
	}
	return l;
}


int upper_bound(vector<int>& nums, int target) {
	int l = 0, r = nums.size(), mid;
	while (l < r)
	{
		mid = (l + r) / 2;
		if (nums[mid] > target)
		{
			r = mid;
		}
		else
		{
			l = mid + 1;
		}
	}
	return l;
}


vector<int> searchRange(vector<int>& nums, int target) {
	if (nums.empty()) return vector<int>{-1, -1};

	int lower = lower_bound(nums, target);
	int upper = upper_bound(nums, target) -1;
	if (lower == nums.size() || nums[lower] != target) {
		return vector<int>({ -1,-1 });
	}

	return vector<int>{lower, upper};
}


void testserachRange() {

	vector<int> nums({ 5,7,7,8,8,10 });
	vector<int> results = searchRange(nums, 8);
	for (auto result : results)
		cout << "the bound is " << result << endl;

}


//有序数组旋转二分查找
bool rotatevectorsearch(vector<int>& nums, int target) {
	int l = 0, r = nums.size()-1, mid;
	while (l<r)
	{
		mid = (l + r) / 2;
		if (nums[mid] == target) {
			return true;
		}
		if (nums[l] == nums[mid])
			++l;
		else if (nums[l]<nums[mid])
		{
			if (target > nums[l] && target < nums[mid])
				r = mid - 1;
			else
				l = mid + 1;

		}
		else if (nums[mid] < nums[r])
			{
				if (target > nums[mid] && target < nums[r])
		  
			        l = mid + 1;
	
			    else
			    	r = mid - 1;

		}
	}

	return false;
}


void testRotateVector() {
	vector<int> nums({2,5,6,0,0,1,2});
	bool result = rotatevectorsearch(nums, 10);
	cout << "the result is " << result << endl;
}

int add(int x, int y) {
	return x + y;
}

int callback(int x, int y, int(*func)(int, int)) {
	return (*func)(x, y);
}

;
__global__ void normkernel(float* src, float* dst, int nums) {
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < nums) {
		//cout << src[idx] << endl;
		//dst[idx] = src[idx];
		dst[idx] = src[idx * 3];
		dst[nums + idx] = src[idx * 3 + 1];
		dst[2 * nums + idx] = src[idx * 3 + 2];
		printf("the dst valuse is %f \n", dst[idx]);
	}
}

void testnormkernel() {
	hipError_t err = hipSuccess;
	int numElements = 20*20*3;
	size_t size = numElements * sizeof(float);
	printf("[Vector addition of %d elements]\n", numElements);

	// Allocate the host input vector A
	float* h_A = (float*)malloc(size);

	// Allocate the host input vector B
	float* h_B = (float*)malloc(size);



	for (int i = 0; i < numElements; ++i) {
		h_A[i] = float(i);
	}

	float* d_A = NULL;
	err = hipMalloc((void**)&d_A, size);

	float* d_B = NULL;
	err = hipMalloc((void**)&d_B, size);

	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	

	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    normkernel <<<blocksPerGrid, threadsPerBlock >>> (d_A, d_B, numElements/3);
	//normkernel << <21, 256 >> > (d_A, d_B, numElements);
	err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
	for (int i = 0; i < numElements; ++i) {
		cout<<h_B[i]<<endl;
	}
	err = hipFree(d_A);
	err = hipFree(d_B);

	free(h_A);
	free(h_B);

}


void add1(int a, int b, int& c, int thread) {
	c = a + b;
}

int main(void) {

	//static const int threads_nums = 10;
	//std::thread threads[threads_nums];
	//int c[threads_nums];
	//for (int i = 0; i < threads_nums; i++)
	//	threads[i] = std::thread(add1, i, i, c[i], i);
	//for (int i = 0; i < threads_nums; i++)
	//	threads[i].join();..



	testnormkernel();


	cin.get();
	//myfirstkernel<<<4, 4>>>();
	////printf("Hello CUDA \n");
	//int h_c;
	//int* d_c;
	//cudaMalloc((void**)&d_c, sizeof(int));
	//gpuAdd << <4, 4 >> > (1,4,d_c);
	//cudaMemcpy(&h_c, d_c, sizeof(int), cudaMemcpyDeviceToHost);
	//printf("the hc is %d", h_c);
	//cudaFree(d_c);

	testcandey();
	testeraseOverlapIntervals();
	testtwosum();
	testmerge();
	testdetectCircle();
	testminwindow();
	testsqrt();
	testserachRange();
	testRotateVector();


	

	return 0;
}